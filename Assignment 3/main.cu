#include "task01.cu.h"
#include "task02.cu.h"
#include "task03.cu.h"
#include "seq.h"

#define ROWS 8192
#define COLS 8192
#define TILE 32
int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1){
	unsigned int resolution=1000000;
	long int diff = (t2->tv_usec + resolution * t2->tv_sec) - (t1->tv_usec + resolution * t1->tv_sec);
	result->tv_sec = diff / resolution;
	result->tv_usec = diff % resolution;
	return (diff<0);
}


void validate_trans(float* mat1, float* mat2){
    int bo = 0;    
    for(int i=0; i<ROWS; i++){
        for(int j=0; j<COLS; j++){
            if (mat1[i*COLS+j] != mat2[j*ROWS+i] ){
                bo = 1;
            } 
        }   
    }
    if(bo == 1){
        printf("validate of transpose failed");
    }
    printf("validate of transpose correct");
}
int main(){
    size_t size = COLS * ROWS;
	size_t mem_size = sizeof(float) * size;
	float* h_A = (float*) malloc(mem_size);
	float* h_B = (float*) malloc(mem_size);
	float* h_C = (float*) malloc(mem_size);

    int const T = 32;
    int dimx = (COLS+T-1)/T;
    int dimy = (ROWS+T-1)/T;
    dim3 block(T,T,1), grid(dimx, dimy, 1);

    unsigned long int elapsed;
    srand(time(0));

    for(int i = 0; i<ROWS; i++){
        for(int j = 0; j<COLS; j++){
            h_A[i*ROWS+j] = i;
        }
    }
    struct timeval t_start,t_end,t_diff;
    //Sequential transpose
	gettimeofday(&t_start, NULL);
    seque_transpose(h_A, h_B, ROWS, COLS);
	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end, &t_start);
	elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);	
	printf("Sequential transpose ran in %lu microseconds.\n",elapsed);
    validate_trans(h_A, h_B);

    //Parallel tranpose
    float* d_A;
	float* d_C;
	hipMalloc((void**)&d_A,mem_size);
	hipMalloc((void**)&d_C,mem_size);

    hipMemcpy(d_A, h_A, mem_size, hipMemcpyHostToDevice);
    gettimeofday(&t_start,NULL);
    transpose_kernel_naive<<<grid,block>>>(d_A, d_C, ROWS, COLS);
	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end, &t_start);
	elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
    printf("Naive Parallel Transpose Kernel ran in %lu microseconds.\n",elapsed);   
    validate_trans(d_A, d_C);
    hipFree(d_A);    
    hipFree(d_C);


	hipMalloc((void**)&d_A,mem_size);
	hipMalloc((void**)&d_C,mem_size);
    hipMemcpy(d_A, h_A, mem_size, hipMemcpyHostToDevice);
    gettimeofday(&t_start,NULL);
    tiling_transpose_kernel<T, TILE><<<grid,block>>>(d_A, d_C, ROWS, COLS);
	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end, &t_start);
	elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
    printf("Tiled Parallel Transpose Kernel ran in %lu microseconds.\n",elapsed);
    hipFree(d_A);    
    hipFree(d_C);

    //Task 2

	hipMalloc((void**)&d_A,mem_size);
	hipMalloc((void**)&d_C,mem_size);
    hipMemcpy(d_A, h_A, mem_size, hipMemcpyHostToDevice);
    gettimeofday(&t_start,NULL);
    quick_2c_kernel<<<grid, block>>>(d_A, d_C, ROWS, COLS);
	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end, &t_start);
	elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
    printf("2.c Kernel ran in %lu microseconds.\n",elapsed);
    hipFree(d_A);    
    hipFree(d_C);

	hipMalloc((void**)&d_A,mem_size);
	hipMalloc((void**)&d_C,mem_size);
    hipMemcpy(d_A, h_A, mem_size, hipMemcpyHostToDevice);
    gettimeofday(&t_start,NULL);
    quick_2c_kernel(d_A, d_C, ROWS);
	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end, &t_start);
	elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
    printf("2.d Kernel ran in %lu microseconds.\n",elapsed);
    hipFree(d_A);    
    hipFree(d_C);

    //Matrix Multiplication

	gettimeofday(&t_start, NULL);    
    matrixmult_seq(h_A, h_A, h_B, ROWS, COLS, COLS);
	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end, &t_start);
	elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
    printf("Sequential Matrix Multiplaction ran in %lu microseconds.\n",elapsed);

    //Parallel Matrix Multiplication
	hipMalloc((void**)&d_A,mem_size);
	hipMalloc((void**)&d_C,mem_size);
    hipMemcpy(d_A, h_A, mem_size, hipMemcpyHostToDevice);
    gettimeofday(&t_start,NULL);
    naive_matrixmult<T><<<grid,block>>>(d_A, d_A, d_C, ROWS, COLS, COLS);
	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end, &t_start);
	elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
    printf("Naive Matrix Multiplication Kernel ran in %lu microseconds.\n",elapsed);
    hipFree(d_A);    
    hipFree(d_C);


}
