#include "task01.cu.h"
#include "task02.cu.h"
#include "task03.cu.h"
#include "seq.h"

#define ROWS 8192
#define COLS 8192
#define TILE 32

int main(){
size_t size = COLS * ROWS;
	size_t mem_size = sizeof(float) * size;
	float* h_A = (float*) malloc(mem_size);
	float* h_B = (float*) malloc(mem_size);
	float* h_C = (float*) malloc(mem_size);

    float* d_A;
	float* d_C;
	hipMalloc((void**)&d_A,mem_size);
	hipMalloc((void**)&d_C,mem_size);

    srand(time(0));
   init_matrix(h_A,size);
}
