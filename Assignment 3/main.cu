#include "task01.cu.h"
#include "task02.cu.h"
#include "task03.cu.h"
#include "seq.h"

#define ROWS 8192
#define COLS 8192
#define TILE 32

int main(){
    size_t size = COLS * ROWS;
	size_t mem_size = sizeof(float) * size;
	float* h_A = (float*) malloc(mem_size);
	float* h_B = (float*) malloc(mem_size);
	float* h_C = (float*) malloc(mem_size);

    int const T = 32;
    int dimx = (COLS+T-1)/T;
    int dimy = (ROWS+T-1)/T;
    dim3 block(T,T,1), grid(dimx, dimy, 1);

    unsigned long int elapsed;
    srand(time(0));

    for(int i = 0; i<ROWS; i++){
        for(int j = 0; j<COLS; j++){
            h_A[i*ROWS+j] = i;
        }
    }
    struct timeval t_start,t_end,t_diff;
    //Sequential transpose
	gettimeofday(&t_start, NULL);
    seque_transpose(h_A, h_B, ROWS, COLS);
	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end, &t_start);
	elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);	
	printf("Sequential transpose ran in %lu microseconds.\n",elapsed);

    //Parallel tranpose
    float* d_A;
	float* d_C;
	hipMalloc((void**)&d_A,mem_size);
	hipMalloc((void**)&d_C,mem_size);

    hipMemcpy(d_A, h_A, mem_size, hipMemcpyHostToDevice);
    gettimeofday(&t_start,NULL);
    transpose_kernel_naive<<<grid,block>>>(d_A, d_C, ROWS, COLS);
	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end, &t_start);
	elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
    printf("Naive Parallel Transpose Kernel ran in %lu microseconds.\n",elapsed);
    hipFree(d_A);    
    hipFree(d_C);


	hipMalloc((void**)&d_A,mem_size);
	hipMalloc((void**)&d_C,mem_size);
    hipMemcpy(d_A, h_A, mem_size, hipMemcpyHostToDevice);
    gettimeofday(&t_start,NULL);
    tiling_transpose_kernel<T, TILE><<<grid,block>>>(d_A, d_C, ROWS, COLS);
	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end, &t_start);
	elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
    printf("Tiled Parallel Transpose Kernel ran in %lu microseconds.\n",elapsed);
    hipFree(d_A);    
    hipFree(d_C);

}
